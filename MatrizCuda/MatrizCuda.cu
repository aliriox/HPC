#include<stdio.h>
#include<stdlib.h>
#include<malloc.h>
#include<time.h>
#include<hip/hip_runtime.h>
#include<string.h>

__global__
void multiplicationKernell(float* m1, float* m2, float* m3, int rowsM1, int colsM2)
{
    int Row = blockIdx.y * blockDim.y + threadIdx.y;
    int Col = blockIdx.x * blockDim.x + threadIdx.x;

    if((Row < rowsM1) && (Col < colsM2))
    {
        float resul = 0.0;
        for(int i = 0; i < rowsM1; i++)
        {
            resul = resul + m1[Row*rowsM1+i] * m2[i*rowsM1+Col];
        }
        m3[Row*rowsM1+Col] = resul;
    }

}

__host__
void toMatrix(float *M, FILE *content, int rows, int cols)
{
    for(int i=0; i<rows;i++)
    {
        for(int j=0; j<cols; j++)
        {
            fscanf(content,"%f",&M[i*cols+j]);
        }
    }
    fclose(content);
}

__host__
void print(float *M, int rows, int cols)
{
    printf("----------MATRIX----------\n");
    for(int i=0; i<rows;i++)
    {
        for(int j=0; j<cols; j++)
        {
            printf("[%f]",M[i*cols+j]);
        }
        printf("\n");
    }
}


int main(int argc, char** argv)
{
    if(argc != 3)
    {
        printf("Error, no se encontraron todos los parametros necesarios.");
        return 1;
    }

    FILE *inputMatrix1;
    FILE *inputMatrix2;

    inputMatrix1 = fopen(argv[1],"r");
    inputMatrix2 = fopen(argv[2],"r");

    float *m1, *m2, *m3;
    int rowsM1, rowsM2, colsM1, colsM2, rowsM3, colsM3;
    
    fscanf(inputMatrix1,"%d",&rowsM1);
    fscanf(inputMatrix1,"%d",&colsM1);

    fscanf(inputMatrix2,"%d",&rowsM2);
    fscanf(inputMatrix2,"%d",&colsM2);

    m1 = (float*) malloc(rowsM1*colsM1*sizeof(float));
    m2 = (float*) malloc(rowsM2*colsM2*sizeof(float));
    m3 = (float*) malloc(rowsM1*colsM2*sizeof(float));
    
    toMatrix(m1, inputMatrix1, rowsM1, colsM1);
    toMatrix(m2, inputMatrix2, rowsM2, colsM2);
    print(m1, rowsM1, colsM1);
    print(m2, rowsM2, colsM2);

    if((rowsM1 != colsM2))
    {
        printf("Error los tamaños de las matrices no son compatibles.");
        return 1;
    }

    //Para el Devince
    hipError_t error = hipSuccess;
    float *d_m1, *d_m2, *d_m3;
    int blockSize = 32;

    dim3 dimBlockSize(blockSize,blockSize,1);
    dim3 dimGridSize(ceil(colsM1 / float(blockSize)), ceil(rowsM1 / float(blockSize)), 1);

    error = hipMalloc((void**)&d_m1, rowsM1 * colsM1 * sizeof(float));
    if(error != hipSuccess)
    {
        printf("Imposible asignar memoria para d_m1");
        return 1;
    }

    error = hipMalloc((void**)&d_m2, rowsM2 * colsM2 * sizeof(float));
    if(error != hipSuccess)
    {
        printf("Imposible asignar memoria para d_m2");
        return 1;
    }

    error = hipMalloc((void**)&d_m3, rowsM3 * colsM3 * sizeof(float));
    if(error != hipSuccess)
    {
        printf("Imposible asignar memoria para d_m3");
        return 1;
    }

    hipMemcpy(d_m1, m1, rowsM1 * colsM1 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_m2, m2, rowsM2 * colsM2 * sizeof(float), hipMemcpyHostToDevice);

    multiplicationKernell<<<dimGridSize, dimBlockSize>>>(d_m1, d_m2, d_m3, rowsM1, colsM2);


    hipMemcpy(m3, d_m3, rowsM1 * colsM2 * sizeof(float), hipMemcpyDeviceToHost);
    print(m3, rowsM1, colsM2);

    free(m1);
    free(m2);
    free(m3);

    hipFree(d_m1);
    hipFree(d_m2);
    hipFree(d_m3);

    return 0;
}