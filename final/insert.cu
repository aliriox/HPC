
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>

using namespace std;

__global__ void insert(int *a, int t){

	int i = blockIdx.x*blockDim.x+threadIdx.x;

	if(i < t-1){
		for(int k = 0; k < t-1; k++){
			if(a[k] > a[k+1]){
				int aux = a[k];
				a[k] = a[k+1];
				a[k+1] = aux;
			}
		}
	}

}

void cuda(int *a, int n){

	int *array;

	array = (int*)malloc(n * sizeof(int));
	hipMemcpy(array,a,n * sizeof(int),hipMemcpyHostToDevice);

	insert<<<1024,1>>>(array,n);

	hipMemcpy(a,array,n * sizeof(int),hipMemcpyDeviceToHost);

	free(array);

}

int main(){

	int n;
	cin >> n;

	int a[n];

	for(int i = 0; i < n; i++){
		cin >> a[i];
	}

	cuda(a,n);

	for(int i = 0; i < n; i++){
		cout << a[i] << " ";
	}
	cout<<endl<<endl;

	return 0;
}
