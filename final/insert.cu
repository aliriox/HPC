
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>

using namespace std;

__global__ void insert(int *a, int t){

	int i = blockIdx.x*blockDim.x+threadIdx.x;

	if(i < t-1){
		for(int k = 0; k < t-1; k++){
			if(a[k] > a[k+1]){
				int aux = a[k];
				a[k] = a[k+1];
				a[k+1] = aux;
			}
		}
	}

}

void imp(int *a, int n){
	for(int i = 0; i < n; i++){
		cout << a[i]<<endl;
	}
	cout<<endl<<endl;
}

void cuda(int *a, int n){

	int *array;

	hipMalloc((void**)&array, n * sizeof(int));
	hipMemcpy(array,a,n * sizeof(int),hipMemcpyHostToDevice);

	cout<< "entro al kernel" <<endl;

	insert<<<1024,1>>>(array,n);

	cout<<"salgo del kernel"<<endl;
	
	hipMemcpy(a,array,n * sizeof(int),hipMemcpyDeviceToHost);
	
	cout<<"libero memoria"<<endl;
	
	hipFree(array);

}

int main(){

	int n;
	cin >> n;

	int a[n];
	int *vec;
        vec = (int*)malloc(n*sizeof(int));

	for(int i = 0; i < n; i++){
		cin >> a[i];
		vec[i] = a[i];
	}
	
	cuda(vec,n);

	imp(vec,n);

	free(vec);
	return 0;
}
