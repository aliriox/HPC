#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>

using namespace std;

__global__ void insert(int *a, int t){

	int i = blockIdx.x*blockDim.x+threadIdx.x;

	__shared__ int tmp[t];
	tmp[i] = a[i];

	if(i < t-1){
		for(int k = 0; k < t-1; k++){
			if(tmp[k] > tmp[k+1]){
				int aux = tmp[k];
				tmp[k] = tmp[k+1];
				tmp[k+1] = aux;
			}
		}
	}

}

void cuda(int *a, int n){

	int *array;

	array = (int*)malloc(n * sizeof(int));
	hipMemcpy(array,a,n * sizeof(int),hipMemcpyHostToDevice);

	insert<<<1024,1>>>(array,n);

	hipMemcpy(a,array,n * sizeof(int),hipMemcpyDeviceToHost);

	free(array);

}

int main(){

	int n;
	cin >> n;

	int a[n];

	for(int i = 0; i < n; i++){
		cin >> a[i];
	}

	cuda(a,n);

	for(int i = 0; i < n; i++){
		cout << a[i] << " ";
	}
	cout<<endl<<endl;

	return 0;
}
