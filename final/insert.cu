
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>

using namespace std;

__global__ void insert(int *a, int t){

	int i = blockIdx.x*blockDim.x+threadIdx.x;

	int cont = 0;

	while(cont <= i*10000){
		cont++;
	}

	if(i < t){
		for(int k = 0; k < t-1; k++){
			if(a[k] > a[k+1]){
				int aux = a[k];
				a[k] = a[k+1];
				a[k+1] = aux;
			}
		}
	}
}

void imp(int *a, int n){
	for(int i = 0; i < n; i++){
		cout << a[i]<<endl;
	}
	cout<<endl<<endl;
}

void cuda(int *a, int n){

	int *array;

	hipMalloc((void**)&array, n * sizeof(int));
	hipMemcpy(array,a,n * sizeof(int),hipMemcpyHostToDevice);

	insert<<<1024,1>>>(array,n);
	
	hipMemcpy(a,array,n * sizeof(int),hipMemcpyDeviceToHost);
	
	hipFree(array);

}

int main(){

	int n;
	cin >> n;

	int a[n];
	int *vec;
        vec = (int*)malloc(n*sizeof(int));

	for(int i = 0; i < n; i++){
		cin >> a[i];
		vec[i] = a[i];
	}
	
	cuda(vec,n);

	imp(vec,n);

	free(vec);
	return 0;
}
