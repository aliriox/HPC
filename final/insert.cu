
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>

using namespace std;

__global__ void insert(int *a, int t){

	int i = blockIdx.x*blockDim.x+threadIdx.x;

	if(i < t-1){
		if(a[i] > a[i+1]){
			int aux = a[i];
			a[i] = a[i+1];
			a[i+1] = aux;
		}
	}
}

void imp(int *a, int n){
	for(int i = 0; i < n; i++){
		cout << a[i]<<endl;
	}
	cout<<endl<<endl;
}

void cuda(int *a, int n){

	int *array;

	hipMalloc((void**)&array, n * sizeof(int));
	hipMemcpy(array,a,n * sizeof(int),hipMemcpyHostToDevice);

	for(int i = 0; i < n; i++){
		insert<<<1024,1>>>(array,n);
	}
	
	hipMemcpy(a,array,n * sizeof(int),hipMemcpyDeviceToHost);
	
	hipFree(array);

}

int main(){

	int n;
	cin >> n;

	int a[n];
	int *vec;
        vec = (int*)malloc(n*sizeof(int));

	for(int i = 0; i < n; i++){
		cin >> a[i];
		vec[i] = a[i];
	}
	
	cuda(vec,n);

	imp(vec,n);

	free(vec);
	return 0;
}
