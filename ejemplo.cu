#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>

#define n 10

__global__ void KernelGPU(float *d_a){

        int i = threadIdx.y + blockDim.y * blockIdx.y;
        int j = threadIdx.x + blockDim.x * blockIdx.x;

        if(i < n && j <  n){
                d_a[i*n+j] = d_a[i*n+j] * 5.0;
        }


}


void createVector(float *a){

        // definiendo variables y reservando memoria

        float *d_a;
        int blocksize = 10;

        hipMalloc((void**)&d_a,n * n * sizeof(float));
        hipMemcpy(d_a, a, n * n * sizeof(float), hipMemcpyHostToDevice);

        dim3 dimBlock(n , n , 1);
	dim3 dimGrid(ceil(n/float(blocksize)),ceil(n/float(blocksize)),1);


        KernelGPU<<< dimGrid, dimBlock>>>(d_a);

        // funcion para liberar memoria
        hipMemcpy(a,d_a, n * n * sizeof(float), hipMemcpyDeviceToHost);

        hipFree(d_a);

}

int main(){

        float *a;

	a = (float *)malloc(n * n *sizeof(float));

        for(int i = 0; i < n * n; i++){
                a[i] = 1.0;
        }

        createVector(a);

        for(int i = 0; i < n; i++){
                for(int j = 0; j < n; j++){
                        printf("[%f]",a[i*n+j]);
                }
                printf("\n");
        }

        return 0;

}


