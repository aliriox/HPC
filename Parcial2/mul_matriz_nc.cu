#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "archivos_csv.c"
#include <time.h>


__global__ void KernelGPU(float *d_a, float *d_b, float *d_c,int f1, int M, int c2){

	int i = threadIdx.y + blockDim.y * blockIdx.y;
	int j = threadIdx.x + blockDim.x * blockIdx.x;
	
	if(i < f1 && j <  c2){
		float Pvalue = 0.0;
		for(int k = 0; k < M; k++){
			Pvalue += d_a[i*M+k] * d_b[k*c2+j];
		}
		d_c[i * c2 + j] = Pvalue;
	}
}


void createVector(float *a, int f1, float *b, int c2, int inteM, float *c){

	// definiendo variables y reservando memoria

	float *d_a, *d_b, *d_c;
	int blocksize = 32;	

	hipMalloc((void**)&d_a, f1 * inteM * sizeof(float));
	hipMemcpy(d_a, a, f1 * inteM * sizeof(float), hipMemcpyHostToDevice);

	hipMalloc((void**)&d_b, inteM * c2 * sizeof(float));
	hipMemcpy(d_b, b, inteM * c2 * sizeof(float), hipMemcpyHostToDevice);

	hipMalloc((void**)&d_c, f1 * c2 * sizeof(float));

	dim3 dimBlock(blocksize , blocksize , 1);
	dim3 dimGrid(ceil(inteM/float(blocksize)),ceil(f1/float(blocksize)),1);


	KernelGPU<<< dimGrid, dimBlock>>>(d_a,d_b,d_c,f1,inteM,c2);

	hipMemcpy(c,d_c, f1 * c2 * sizeof(float), hipMemcpyDeviceToHost);
	
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

}

int main(int argc, char *argv[]){

	if(argc != 3){
		printf("no se han ingresado los archivos necesarios\n");
	}

	FILE *fp;
	fp = fopen(argv[1], "r");

	if(fp==NULL){
		fputs("File error",stderr);
		return 1;
	}
	fclose(fp);
	
	fp = fopen(argv[2], "r");

	if(fp == NULL){
		fputs("file error", stderr);
		return 1;
	}
	fclose(fp);

	int Rows_1 = Detected_rows(fp,argv[1]);
	int Columns_1 = Detected_columns(fp,argv[1]);

	int Rows_2 = Detected_rows(fp,argv[2]);
	int Columns_2 = Detected_columns(fp,argv[2]);

	if(Columns_1 != Rows_2){
		printf("las matrizes no cumplen los requisitos para la multiplicacion entre estas");
		return 1;
	}

	int inteM = Columns_1;	
	
	float *Matriz1 = (float*)malloc(Rows_1 * inteM * sizeof(float));
	float *Matriz2 = (float*)malloc(inteM * Columns_2 * sizeof(float));
	float *MatrizR = (float*)malloc(Rows_1 * Columns_2 * sizeof(float));

	ExtracData(fp, Matriz1,argv[1],Rows_1,inteM);
	ExtracData(fp, Matriz2,argv[2],inteM,Columns_2);
	
	printf("Matriz 1: \n");
	printMatriz(Matriz1,Rows_1, inteM);
	printf("\n");
	printf("matriz 2: \n");
	printMatriz(Matriz2,inteM,Columns_2);
	printf("\n");

	time_inicial = clock();
	createVector(Matriz1,Rows_1,Matriz2,Columns_2,inteM,MatrizR);
	time_final = clock();

	printf("tiempo de ejecucion: %f\n", (time_final-time_inicial/CLOCKS_PER_SEC));

	printf("matriz Respuesta: \n");
	printMatriz(MatrizR, Rows_1, Columns_2);
	printf("\n");

	free(Matriz1);
	free(Matriz2);
	free(MatrizR);	

	return 0;

}
